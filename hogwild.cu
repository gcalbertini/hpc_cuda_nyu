#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "utils.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define threadsperblock 1000

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

template <typename T>
void swap(T *a, T *b)
{
    T temp = *a;
    *a = *b;
    *b = temp;
}

// Shuffle x and y in the same way, x is n * k, y is n
// SGD requires random selection for mini batches. Random shuffle the whole
// train data and looping from the start serves the same purpose of random selection
template <typename T>
void shuffleXY(T x, T y, size_t n, size_t k)
{
    if (n == 0)
        return;
    srand((unsigned)time(NULL));
    for (size_t i = n - 1; i > 0; --i)
    {
        size_t j = rand() % (i + 1);
        swap(y + i, y + j);
        for (size_t kk = 0; kk < k; kk++)
        {
            swap(x + i * k + kk, x + j * k + kk);
        }
    }
}

double *train_x_csv()
{
    std::ifstream f;
    std::string line; /* string for line & value */
    long nrows = 0;
    long ncols = 0;

    f.open("generated_data/df_X.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    std::stringstream lineStream;
    std::string lastline;
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << nrows++
        //           << " lineStream.str() = " << lineStream.str() << std::endl;
        nrows++;
    }

    // just reads last line to count columns just by counting commas+1
    while (std::getline(lineStream, lastline, ','))
    {
        // std::cout << "cell=" << lastline << std::endl;
        ++ncols;
    }
    f.close();

    f.open("generated_data/df_X.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    // std::cout << ncols << std::endl;
    double *train_x = (double *)aligned_malloc(ncols * nrows * sizeof(double));
    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            train_x[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();

    return train_x;
}

double *train_y_csv()
{
    std::ifstream f;
    std::string line; /* string for line & value */
    long nrows = 0;

    f.open("generated_data/df_y.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    std::stringstream lineStream;
    std::string lastline;
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << nrows++
        //           << " lineStream.str() = " << lineStream.str() << std::endl;
        nrows++;
    }
    f.close();

    f.open("generated_data/df_y.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    double *train_y = (double *)aligned_malloc(nrows * sizeof(double));
    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            train_y[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();

    return train_y;
}

__global__
void hogwild_kernel(int num_epochs, long train_size, long numpredictors, int batch_size, double learning_rate ,double *X, double *y, double *weights, double *w_gradients, double *pred) {
    double b_gradient = 0;

    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int epoch = 0; epoch < num_epochs; epoch++)
    {
        long start = batch_size*idx;
        long pred_start_index = idx*batch_size;
        for (long i = start; i < start + batch_size; i++)
        {

            weights[0] = weights[0] - (b_gradient / batch_size) * learning_rate;
            for (long k = 0; k < numpredictors; k++)
            {
                weights[k + 1] = weights[k + 1] - (w_gradients[idx*numpredictors+k] / batch_size) * learning_rate;
            }

            // update prediction using the new weights
            // y = a + b*(x_0) + c*(x_1)^2 + d*(x_2)^3 + ....
            // a, b, c, d ... are weights, x_0, x_1, x_2 are predictor_values
                
            double pred_reduction_sum = weights[0];
            for (long j = 0; j < numpredictors; j++)
            {
                pred_reduction_sum += weights[j + 1] * pow(X[i * numpredictors + j], j + 1);
            }
            pred[pred_start_index+i-start] = pred_reduction_sum;
            // loss += pow(pred_reduction_sum - train_y[i], 2);
                
        }

            
        for (long i = start; i < start + batch_size; i++)
        {
            for (long k = 1; k <= numpredictors; k++)
            {
                w_gradients[idx*numpredictors+k] += 2 * (pred[pred_start_index+i-start] - y[i]) * (k * weights[k] * pow(X[i * numpredictors + k], k - 1));
            
            }
            b_gradient += 2 * (pred[pred_start_index+i-start] - y[i]);
        }
    }
    
}

int main(int argc, char * argv[])
{
    long numpredictors;
    int batch_size;
    long train_size;
    int num_epochs;
    double learning_rate = 0.05;

    if(argc != 5)
    {
        fprintf(stderr, "usage: hogwild train_size numpredictors batch_size num_epochs\n");
        fprintf(stderr, "train_size = number of data points\n");
        fprintf(stderr, "numpredictors = number of predictors for each data point\n");
        fprintf(stderr, "batch_size = number of data points in each batch\n");
        fprintf(stderr, "num_epochs = number of epochs for training\n");
        exit(1);
    }

    train_size = atol(argv[1]);
    numpredictors = atol(argv[2]);
    batch_size = atoi(argv[3]);
    num_epochs = atoi(argv[4]);

    int numblocks;
    int total_threads = train_size / batch_size;

    if (train_size % batch_size != 0) {
        total_threads += 1;
    }
 
    if( (total_threads % threadsperblock) == 0 )
	    numblocks = total_threads/ threadsperblock;
    else 
      	numblocks = (total_threads/threadsperblock)>0? (total_threads/threadsperblock)+1:1 ;

    //X, y comes from csv function now? Both now should be C array
    double *X = train_x_csv();
    double *y = train_y_csv();
    // shuffleXY(X,y,train_size,numpredictors)

   double *weights = (double *)malloc(sizeof(double) * (numpredictors + 1));
    memset(weights, 0, numpredictors);

    double *weights_d, *w_gradients_d, *pred_d, *X_d, *y_d ;

    size_t wg_size = size_t(total_threads*numpredictors) * sizeof(double);
    size_t pred_size = size_t(total_threads*batch_size) * sizeof(double);

    checkCuda(hipMalloc((void**)&w_gradients_d, wg_size));
    checkCuda(hipMemset(w_gradients_d, 0, wg_size));
    checkCuda(hipMalloc((void**)&pred_d, pred_size));
    // checkCuda(hipMemset(pred_d, 0, pred_size));

    checkCuda(hipMalloc((void**)&weights_d, (numpredictors+1)*sizeof(double)));
    checkCuda(hipMalloc((void**)&X_d, train_size*(numpredictors)*sizeof(double)));
    checkCuda(hipMalloc((void**)&y_d, train_size*sizeof(double)));

    // assume weights and prediction are initialized
    checkCuda(hipMemcpyAsync(weights_d, weights, (numpredictors+1)*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(X_d, X, train_size*(numpredictors)*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(y_d, y, train_size*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());

    printf("GPU: %d blocks of %d threads each\n", numblocks, threadsperblock); 

    hogwild_kernel<<<numblocks , threadsperblock>>>(num_epochs, train_size, numpredictors, batch_size, learning_rate, X_d, y_d, weights_d, w_gradients_d, pred_d);


    free(weights);
    //free(pred);

    hipFree(weights_d);
    //hipFree(pred_d);

    return 0;

}

