#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "utils.h"

template <typename T>
void swap(T *a, T *b)
{
    T temp = *a;
    *a = *b;
    *b = temp;
}

// Shuffle x and y in the same way, x is n * k, y is n
// SGD requires random selection for mini batches. Random shuffle the whole
// train data and looping from the start serves the same purpose of random selection
template <typename T>
void shuffleXY(T x, T y, size_t n, size_t k)
{
    if (n == 0)
        return;
    srand((unsigned)time(NULL));
    for (size_t i = n - 1; i > 0; --i)
    {
        size_t j = rand() % (i + 1);
        swap(y + i, y + j);
        for (size_t kk = 0; kk < k; kk++)
        {
            swap(x + i * k + kk, x + j * k + kk);
        }
    }
}

// Assume equation of form y= a + bx + cx^2 + dx^3 + ....
// Squared L2 norm
// Loss for single example = (y* - y)^2
// Derivative of loss wrt coeff_i = 2(y* - y)*(i*coeffs[i]*predictor_values[i]^(i-1))

double difflinear(double *weights, double predictor_value, long desired_coeff, double pred, double train_y)
{
    if (desired_coeff == 0)
        return 2 * (pred - train_y);

    return 2 * (pred - train_y) * (desired_coeff * weights[desired_coeff] * pow(predictor_value, desired_coeff - 1));
}

// calculate gradient for each batch
void calc_gradient(double *train_x, double *train_y, int batch_size, long numpredictors, double *weights, double *pred, double *w_gradients, double *b_gradient)
{

    for (int b = 0; b < batch_size; b++)
    {
        for (long k = 1; k <= numpredictors; k++)
        {
            w_gradients[k] += difflinear(weights, train_x[b * numpredictors + k], k, pred[b], train_y[b]);
        }
        b_gradient[0] += difflinear(weights, train_x[0], 0, pred[b], train_y[b]);
    }
}

void calc_pred(double *train_x, double *weights, int batch_size, double *w_gradients, double *b_gradient, long numpredictors, double learning_rate, double *pred)
{

    // calculate the actual prediction using the gradients.
    // Calculate it in mini-batch gradients and then update

    // update weights based using the gradients
    weights[0] = weights[0] - (b_gradient[0] / batch_size) * learning_rate;
    for (long i = 0; i < numpredictors; i++)
    {
        weights[i + 1] = weights[i + 1] - (w_gradients[i] / batch_size) * learning_rate;
    }

    // update prediction using the new weights
    // y = a + b*(x_0) + c*(x_1)^2 + d*(x_2)^3 + ....
    // a, b, c, d ... are weights, x_0, x_1, x_2 are predictor_values
    for (long i = 0; i < batch_size; i++)
    {
        double pred_reduction_sum = weights[0];
        for (long j = 0; j < numpredictors; j++)
        {
            pred_reduction_sum += weights[j + 1] * pow(train_x[i * numpredictors + j], j + 1);
        }
        pred[i] = pred_reduction_sum;
    }
}

double *train_x_csv()
{
    std::ifstream f;
    std::string line; /* string for line & value */
    long nrows = 0;
    long ncols = 0;

    f.open("generated_data/df_X.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    std::stringstream lineStream;
    std::string lastline;
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << nrows++
        //           << " lineStream.str() = " << lineStream.str() << std::endl;
        nrows++;
    }

    // just reads last line to count columns just by counting commas+1
    while (std::getline(lineStream, lastline, ','))
    {
        // std::cout << "cell=" << lastline << std::endl;
        ++ncols;
    }
    f.close();

    f.open("generated_data/df_X.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    // std::cout << ncols << std::endl;
    double *train_x = (double *)aligned_malloc(ncols * nrows * sizeof(double));
    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            train_x[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();

    return train_x;
}

double *train_y_csv()
{
    std::ifstream f;
    std::string line; /* string for line & value */
    long nrows = 0;

    f.open("generated_data/df_y.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    std::stringstream lineStream;
    std::string lastline;
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << nrows++
        //           << " lineStream.str() = " << lineStream.str() << std::endl;
        nrows++;
    }
    f.close();

    f.open("generated_data/df_y.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    double *train_y = (double *)aligned_malloc(nrows * sizeof(double));
    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            train_y[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();

    return train_y;
}

// decide if all double or all type long throughout?
double *train_x_C(const unsigned long batch_size, const unsigned long numpredictors)
{
    /* Generate a new random seed from system time - do this once in your constructor */
    srand(time(0));

    double *train_x = (double *)aligned_malloc(batch_size * numpredictors * sizeof(double));
    for (long i = 0; i < batch_size * numpredictors; i++)
        train_x[i] = 123.4; // drand48();

    return train_x;
}

double *train_y_C(const unsigned long batch_size, const unsigned long numpredictors, double *train_x)
{

    double *train_y = (double *)(aligned_malloc(sizeof *train_y * batch_size));
    // Define random generator with Gaussian distribution
    double b = 123.;
    const double mean = 0.0;
    const double stddev = 0.2;
    std::default_random_engine generator;
    std::normal_distribution<double> dist(mean, stddev);

    // Add Gaussian noise too
    for (long i = 0; i < batch_size; i++)
    {
        for (long j = 0; j < numpredictors; j++)
        {
            train_y[j] = train_x[i * numpredictors + j] + b + dist(generator);
        }
    }

    return train_y;
}

int main(int argc, char *argv[])
{
    long numpredictors;
    int batch_size;
    long train_size;
    int num_epochs;
    double learning_rate = 0.05;

    train_size = atol(argv[1]);
    numpredictors = atol(argv[2]);
    batch_size = atoi(argv[3]);
    num_epochs = atoi(argv[4]);

    //X, y comes from csv function now? Both now should be C array

    /// Assume the above is implemented

    /// todo: initialize random weights and gradients
    double *w_gradients = (double *)malloc(sizeof(double) * numpredictors);
    double *b_gradient = (double *)malloc(sizeof(double));
    double *weights = (double *)malloc(sizeof(double) * (numpredictors + 1));
    ///

    double *train_batch_x = (double *)malloc(sizeof(double) * batch_size * numpredictors);
    double *train_batch_y = (double *)malloc(sizeof(double) * batch_size);
    double *pred = (double *)malloc(sizeof(double) * batch_size);
    long start = 0;

    for (int epoch = 0; epoch < num_epochs; epoch++)
    {
        for (long i = 0; i < train_size; i++)
        {
            for (long j = 0; j < numpredictors; j++)
            {
                train_batch_x[start * numpredictors + j] = X[i * numpredictors + j];
            }
            train_batch_y[start] = y[i];
            start++;
            if ((i + 1) % batch_size == 0)
            {
                calc_pred(train_batch_x, weights, batch_size, w_gradients, b_gradient, numpredictors, learning_rate, pred);
                calc_gradient(train_batch_x, train_batch_y, batch_size, numpredictors, weights, pred, w_gradients, b_gradient);
                start = 0;
            }
        }
    }

    return 0;
}
