#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "utils.h"


// Assume equation of form y= a + bx + cx^2 + dx^3 + ....
// Squared L2 norm
// Loss for single example = (y* - y)^2
// Derivative of loss wrt coeff_i = 2(y* - y)*(i*coeffs[i]*predictor_values[i]^(i-1))

float difflinear(long numpredictors, double *coeffs, double *predictor_values, long desired_coeff, double pred, double train_y)
{
    if (desired_coeff == 0)
        return 2 * (pred - train_y);

    return 2 * (pred - train_y) * (desired_coeff * coeffs[desired_coeff] * pow(predictor_values[desired_coeff], desired_coeff - 1));
}

void calc_gradient(double *train_x, double *train_y, double *weights, double *constant, long batch_size, long numpredictors, double *pred)
{
    // Note: have type long and type double mixing but I believe both 8 bytes here so happen to be ok? Best to chanage evertyhing to double or long double? Assuming we compile in C++ as his examples showed with CUDA
    double *temp_x = (double *)(malloc(sizeof *temp_x * numpredictors)); // frees us from having to worry about changing the RHS of the expression if ever change the type of temp_x: https://stackoverflow.com/questions/605845/do-i-cast-the-result-of-malloc
    double *w_gradients = (double *)(malloc(sizeof *w_gradients * numpredictors));
    double b_gradient = 0;

    for (long i = 0; i < batch_size; i++)
    {
        for (long j = 0; j < numpredictors; j++)
        {
            temp_x[j] = train_x[i * numpredictors + j];
        }

        for (long k = 1; k <= numpredictors; k++)
        {
            w_gradients[k] += difflinear(numpredictors, weights, temp_x, k, pred[i], train_y[i]);
        }
        b_gradient += difflinear(numpredictors, weights, temp_x, 0, pred[i], train_y[i]);
    }
}

void calc_pred(double *train_x)
{

    // calculate the actual prediction using the gradients.
    // Calculate it in mini-batch gradients and then update
}

// May want to create namespaces such such that we can use things like GPU::calc_gradient and CPU::calc_pred calc_gradient