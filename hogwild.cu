#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <iterator>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "utils.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define threadsperblock 1000

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

template <typename T>
void swap(T *a, T *b)
{
    T temp = *a;
    *a = *b;
    *b = temp;
}

// Shuffle x and y in the same way, x is n * k, y is n
// SGD requires random selection for mini batches. Random shuffle the whole
// train data and looping from the start serves the same purpose of random selection
template <typename T>
void shuffleXY(T x, T y, size_t n, size_t k)
{
    if (n == 0)
        return;
    srand((unsigned)time(NULL));
    for (size_t i = n - 1; i > 0; --i)
    {
        size_t j = rand() % (i + 1);
        swap(y + i, y + j);
        for (size_t kk = 0; kk < k; kk++)
        {
            swap(x + i * k + kk, x + j * k + kk);
        }
    }
}

void train_x_csv(double *X, long nrows, long ncols)
{
    std::ifstream f;
    std::string line; /* string for line & value */
    std::stringstream lineStream;

    f.open("generated_data/df_X_train.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            X[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();
}

void train_y_csv(double *y, long nrows)
{
    std::ifstream f;
    std::string line; /* string for line & value */
    std::stringstream lineStream;

    f.open("generated_data/df_y_train.csv"); /* open file with filename as argument */
    if (!f.is_open())
    { /* validate file open for reading */
        std::cerr << "error: file open failed!\n";
    }

    long idx = 0;
    // read lines
    while (std::getline(f, line))
    {
        lineStream.clear();
        lineStream.str(line);
        // std::cout << "row=" << row++
        //   << " lineStream.str() = " << lineStream.str() << std::endl;
        while (std::getline(lineStream, line, ','))
        {
            // std::cout << "element=" << line << std::endl;
            y[idx] = atof(line.c_str());
            idx++;
        }
    }
    f.close();
}

__global__
void hogwild_kernel(int num_epochs, int train_size, long numpredictors, int batch_size, double learning_rate ,double *X, double *y, double *weights, double *w_gradients, double *pred, double *loss_arr) {
    double b_gradient = 0;
    double loss;
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int epoch = 0; epoch < num_epochs; epoch++)
    {
        long start = batch_size*idx;
        loss = 0;
        for (long i = start; i < start + batch_size; i++)
        {
            if (i < train_size) 
            {              
                double pred_reduction_sum = weights[0];
                for (long j = 0; j < numpredictors; j++)
                {
                    pred_reduction_sum += weights[j + 1] * pow(X[i * numpredictors + j], j + 1);
                    w_gradients[idx*numpredictors+j] = 0;
                }
                pred[i] = pred_reduction_sum;
                b_gradient = 0;
                loss += pow(pred_reduction_sum - y[i], 2);

                for (long k = 1; k <= numpredictors; k++)
                {
                    w_gradients[idx*numpredictors+(k-1)] += -2 * (y[i] - pred[i]) *  pow(X[i * numpredictors + (k-1)], k);
                
                }
                b_gradient += -2 * (y[i] - pred[i]);

                weights[0] = weights[0] - (b_gradient / batch_size) * learning_rate;
                for (long k = 0; k < numpredictors; k++)
                {
                    weights[k + 1] = weights[k + 1] - (w_gradients[idx*numpredictors+k] / batch_size) * learning_rate;
                    
                }
            }
                
        }
        
        learning_rate = learning_rate / 2;

        loss_arr[idx*epoch+epoch] = loss;
    }
    
}

int main(int argc, char * argv[])
{
    long numpredictors;
    int batch_size;
    long train_size;
    int num_epochs;
    double learning_rate = 0.05;

    if(argc != 5)
    {
        fprintf(stderr, "usage: hogwild train_size numpredictors batch_size num_epochs\n");
        fprintf(stderr, "train_size = number of data points\n");
        fprintf(stderr, "numpredictors = number of predictors for each data point\n");
        fprintf(stderr, "batch_size = number of data points in each batch\n");
        fprintf(stderr, "num_epochs = number of epochs for training\n");
        exit(1);
    }

    train_size = atol(argv[1]);
    numpredictors = atol(argv[2]);
    batch_size = atoi(argv[3]);
    num_epochs = atoi(argv[4]);

    int numblocks;
    int total_threads = train_size / batch_size;

    if (train_size % batch_size != 0) {
        total_threads += 1;
    }
 
    if( (total_threads % threadsperblock) == 0 )
	    numblocks = total_threads/ threadsperblock;
    else 
      	numblocks = (total_threads/threadsperblock)>0? (total_threads/threadsperblock)+1:1 ;

    //X, y comes from csv function now? Both now should be C array
    double *X = (double *)malloc(sizeof(double) * numpredictors * train_size);
    double *y = (double *)malloc(sizeof(double) * train_size);
    train_x_csv(X, train_size, numpredictors);
    train_y_csv(y, train_size);
    // shuffleXY(X,y,train_size,numpredictors)

    double *weights = (double *)malloc(sizeof(double) * (numpredictors + 1));
    double *loss = (double *)malloc(sizeof(double) * num_epochs*total_threads);
    std::fill_n(weights, 0, numpredictors);

    double *weights_d, *w_gradients_d, *pred_d, *X_d, *y_d, *loss_d ;

    size_t wg_size = size_t(total_threads*numpredictors) * sizeof(double);
    size_t pred_size = size_t(total_threads*batch_size) * sizeof(double);
    size_t loss_size = size_t(total_threads*num_epochs) * sizeof(double);

    checkCuda(hipMalloc((void**)&w_gradients_d, wg_size));   
    checkCuda(hipMalloc((void**)&pred_d, pred_size));
    checkCuda(hipMalloc((void**)&loss_d, loss_size));
    checkCuda(hipMemset(w_gradients_d, 0, wg_size));
    checkCuda(hipMemset(pred_d, 0, pred_size));
    checkCuda(hipMemset(loss_d, 0, loss_size));

    checkCuda(hipMalloc((void**)&weights_d, (numpredictors+1)*sizeof(double)));
    checkCuda(hipMalloc((void**)&X_d, train_size*(numpredictors)*sizeof(double)));
    checkCuda(hipMalloc((void**)&y_d, train_size*sizeof(double)));

    // assume weights and prediction are initialized
    checkCuda(hipMemcpyAsync(weights_d, weights, (numpredictors+1)*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(X_d, X, train_size*(numpredictors)*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpyAsync(y_d, y, train_size*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());

    printf("GPU: %d blocks of %d threads each\n", numblocks, threadsperblock); 

    hogwild_kernel<<<numblocks , threadsperblock>>>(num_epochs, train_size, numpredictors, batch_size, learning_rate, X_d, y_d, weights_d, w_gradients_d, pred_d, loss_d);
    checkCuda(hipMemcpyAsync(loss, loss_d, loss_size, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    for(int i=0; i<num_epochs; i++){
        //printf("Epoch: %d Average loss: %f\n", i+1, loss[i] / ((train_size) / batch_size));
        double cur_loss = 0;
        for (int j=0; j<total_threads; j++) {
            cur_loss += loss[j*num_epochs+i];
        }
        printf("Epoch: %d Average loss: %f\n", i+1, cur_loss/total_threads);
    }

    free(weights);
    //free(pred);

    hipFree(weights_d);
    //hipFree(pred_d);

    return 0;

}

